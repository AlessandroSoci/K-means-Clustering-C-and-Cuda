#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <sys/time.h>
#include <sstream>
#include <stdlib.h>
#include <math.h>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime_api.h>
#include "utils.h"


using namespace std;
using namespace cv;

__device__ float Eu_distance(float B, float G, float R, float *centroid, int k)
{
    float sum;
    float dist;
    sum = float(pow(B-centroid[k+0], 2) + pow(G-centroid[k+1], 2) + pow(R-centroid[k+2], 2));
    dist = float(sqrt(sum));
    return dist;
}


__global__ void match_point(int *cluster, float *B_c, float *G_c, float *R_c, float *centr, int size_image, int n_threads, int K)
{
	int i = threadIdx.x;

	int size_per_thread = size_image/n_threads;
    int start = i*size_per_thread;
	int end = start + size_per_thread;

	float distance, distance_old;

	if (i >=size_image){ return; }

	if (i==n_threads-1)
    {
        start = (n_threads-1)*size_per_thread;
        end = size_image;
	}

	for(int j = start; j<end; j++)
	{
		for(int k=0; k<K; k++)
		{
			distance = Eu_distance(B_c[j], G_c[j], R_c[j], centr, k*3);

            if (k == 0)
            {
                distance_old = distance;
                cluster[j] = k;
                
            }
            else if (distance<=distance_old)
            {
                distance_old = distance;
                cluster[j] = k;
            }

		}
	}
}

__global__ void update_cluster(int *cluster, float *centroid, float *B_c, float *G_c, float *R_c, int size_image, int n_threads, int K)
{

	extern __shared__ float sdata[];
	float *nValue = sdata;
	float *Bdata = &nValue[(K)*n_threads-1];
	float *Gdata = &Bdata[(K)*n_threads];
	float *Rdata = &Gdata[(K)*n_threads];

	unsigned int tid = threadIdx.x;
	int k = blockIdx.x;

	int size_per_thread = int(size_image/n_threads);
    int start = tid*size_per_thread;
	int end = start + size_per_thread;

	float count = 0;
	float B = 0;
	float G = 0;
	float R = 0;	

	if (tid >=size_image){ return; }

	if (tid==n_threads-1)
    {
        start = (n_threads-1)*size_per_thread;
        end = size_image;
	}
	for(int j = start; j < end; j++)
	{
		if(cluster[j] == k)
		{
            B = B + (B_c[j]);
            G = G + (G_c[j]);
            R = R + (R_c[j]);
            count = count + 1; 
		}
	}

    nValue[tid] = count;
    Bdata[tid] = B;
    Gdata[tid] = G;
    Rdata[tid] = R;

    __syncthreads();

    for(unsigned int s=1; s < blockDim.x; s *= 2) 
	{
		if(tid % (2*s) == 0)
		{
			nValue[tid] += nValue[tid + s];
			Bdata[tid] += Bdata[tid + s];
			Gdata[tid] += Gdata[tid + s];
			Rdata[tid] += Rdata[tid + s];
		}
		__syncthreads();
	}


	if(tid == 0)
	{
		if (nValue[0] != 0)
		{
			centroid[k*3 + 0] = Bdata[0] / nValue[0];
			centroid[k*3 + 1] = Gdata[0] / nValue[0];
			centroid[k*3 + 2] = Rdata[0] / nValue[0];
		}
	}
}


int main(int argc, char**argv)
{

	clock_t start, end;
	double cpu_time_used;


	char *str_image;
    str_image = argv[1];
    cout<< str_image << endl;

    cv::Mat image;	
    image = cv::imread(str_image , CV_LOAD_IMAGE_COLOR);

    int size_image = (image.rows*image.cols);
    cout << size_image << endl;

    if(! image.data ) {
        std::cout <<  "Could not open or find the image" << std::endl ;
        return -1;
    }

    unsigned int k;
    cout << "Selezionare il numero di Cluster desiderati:"<< endl;
    cin>>k;

    unsigned int n_threads;
    cout << "Selezionare il numero di thread desiderati:"<< endl;
    cin>>n_threads;


    start = clock();

	float *B_channel =  new float [size_image];
	float *G_channel =  new float [size_image];
	float *R_channel =  new float [size_image];

	float *B_c, *G_c, *R_c;

	// vettorizzazione delle coordinate BGR per portarle in __device__
	for(int i=0; i<size_image; i++)
	{
 	   B_channel[i] = float(image.data[image.channels()*i+0]);
       G_channel[i] = float(image.data[image.channels()*i+1]);
       R_channel[i] = float(image.data[image.channels()*i+2]);
	}

	hipMalloc(&B_c, size_image* sizeof(float));
	hipMalloc(&G_c, size_image* sizeof(float));
	hipMalloc(&R_c, size_image* sizeof(float));
	hipMemcpy(B_c, B_channel, size_image* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(G_c, G_channel, size_image* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(R_c, R_channel, size_image* sizeof(float), hipMemcpyHostToDevice);

    // Inizializzazione dei cluster
    float *Centroids = new float[k*3];
    float *centr;
    // Coordinate dei centroidi
    float C_x, C_y, C_z;

    int count = 0;
    // seed per la funzione rand
    // Commentare se si vuole avere sempre gli stessi centroidi
   	//srand(time(NULL));
    for (int j=0; j < k*3; j = j + 3){
        C_x = abs(rand()%255);
        C_y = abs(rand()%255);
        C_z = abs(rand()%255);
        Centroids[j+0] = C_x;
        Centroids[j+1] = C_y;
        Centroids[j+2] = C_z;
    }

    // verifica della posizione dei centroidi
    for(int j=0; j < k*3; j++){
        cout << Centroids[j] << " ";
        if (j%3 == 2){
            cout << " Centroide numero: " << count << "\n";
            count ++;
        }
    }

    hipMalloc(&centr, k*3*sizeof(float));
    hipMemcpy(centr, Centroids, k*3*sizeof(float), hipMemcpyHostToDevice);

    // variabile utile per il confronto dei vecchi centroidi con i nuovi
    float *Centroids_old = new float [k*3];
    // variabile che indica l'appartenenza a quale cluster
    int *Clusters = new int [size_image];
    int *clust;
    hipMalloc(&clust, size_image*sizeof(int));
    // variabile di errore tra il vecchio e il nuoce centroide;
    float error = 100000;

    while (error > 0){

    	match_point<<<1,n_threads>>>(clust, B_c, G_c, R_c, centr, size_image, n_threads, k);
		hipDeviceSynchronize();

		hipMemcpy(Clusters, clust, size_image*sizeof(int), hipMemcpyDeviceToHost);

		// Deep copy
        for(int i=0; i < k*3; i++){
            Centroids_old[i] = Centroids[i];
        }

        // calcolo dei nuovi centroidi
        update_cluster<<<k,n_threads, (k)*n_threads*sizeof(float)+(k)*n_threads*sizeof(float)+(k)*n_threads*sizeof(float)+(k)*n_threads*sizeof(float)>>>(clust, centr, B_c, G_c, R_c, size_image, n_threads, k);
        hipDeviceSynchronize();
        hipMemcpy(Centroids, centr, k*3*sizeof(float), hipMemcpyDeviceToHost);

        for (int j=0; j < k*3; j = j + 3)
        {
            cout << "Centroidi con metodo parallelo: " << Centroids[j+0] << " " <<Centroids[j+1] << " " <<Centroids[j+2] << endl;
        }

        error = error_distance(Centroids, Centroids_old, k);
        cout << error << endl;
        //cin>>count;
    }

    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

    cout << "Tempo di esecuzione: " << cpu_time_used << endl;

    for(int i=0; i<k; i++)
    {
        for (int j = 0; j < size_image; j++) {
            if (Clusters[j] == i){
                image.data[image.channels()*j+0] = uchar(Centroids[i+0]);
                image.data[image.channels()*j+1] = uchar(Centroids[i+1]);
                image.data[image.channels()*j+2] = uchar(Centroids[i+2]);
            }
        }
        cout<<"it's working" << endl;
    }

    resize(image, image, Size(1024, 768), 0, 0, INTER_CUBIC);


    namedWindow( "Display window", WINDOW_AUTOSIZE );    // Create a window for display.
    imshow("Display window", image);                 // Show our image inside it.

    waitKey(0);
    return 0;

}

